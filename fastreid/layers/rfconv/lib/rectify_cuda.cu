#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/div_rtn.h>
#include <ATen/Dispatch.h>
#include <ATen/TensorUtils.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>

#include "utils.h"

template <typename scalar_t, typename accscalar_t>
__global__ void conv_rectify_cuda_frame(
    const int nthreads,
    //const scalar_t* const bottom_data,
    const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w,
    scalar_t* const top_data,
    bool average_mode) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    //const int c = (index / pooled_width / pooled_height) % channels;
    //const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    const int pool_size = ((kernel_h - 1) / dilation_h + 1) * ((kernel_w - 1) / dilation_w + 1);
    //const int pool_size = ((hend - hstart - 1) / dilation_h + 1) * ((wend - wstart - 1) / dilation_w + 1);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    accscalar_t mul_factor;
    int hcount = int(((hend - hstart) - 1) / dilation_h + 1);
    int wcount = int(((wend - wstart) - 1) / dilation_w + 1);
    if (average_mode) {
      mul_factor = accscalar_t(1.0) / (hcount * wcount);
    }
    else {
      mul_factor = accscalar_t(1.0) * pool_size / (hcount * wcount);
    }
    top_data[index] = ScalarConvert<accscalar_t, scalar_t>::to(top_data[index] * mul_factor);
  }
}

void conv_rectify_cuda_tempalte(
  at::Tensor& output,
  const at::Tensor& input_,
  at::IntArrayRef kernel_size,
  at::IntArrayRef stride,
  at::IntArrayRef padding,
  at::IntArrayRef dilation,
  bool average)
{
  //at::TensorArg output_arg{ output, "output", 1 };
  //at::TensorArg input_arg{ input_, "input_", 2 };

  //checkAllSameGPU("rectify_out_cuda", {output_arg, input_arg});

  // #20866, #22032: Guarantee this for the official C++ API?
  TORCH_CHECK(kernel_size.size() == 1 || kernel_size.size() == 2,
    "rectify: kernel_size must either be a single int, or a tuple of two ints");
  const int kH = safe_downcast<int, int64_t>(kernel_size[0]);
  const int kW = kernel_size.size() == 1 ? kH : safe_downcast<int, int64_t>(kernel_size[1]);

  TORCH_CHECK(stride.empty() || stride.size() == 1 || stride.size() == 2,
    "rectify: stride must either be omitted, a single int, or a tuple of two ints");
  const int dH = stride.empty() ? kH : safe_downcast<int, int64_t>(stride[0]);
  const int dW = stride.empty() ? kW :
                 stride.size() == 1 ? dH : safe_downcast<int, int64_t>(stride[1]);

  TORCH_CHECK(padding.size() == 1 || padding.size() == 2,
    "rectify: padding must either be a single int, or a tuple of two ints");
  const int padH = safe_downcast<int, int64_t>(padding[0]);
  const int padW = padding.size() == 1 ? padH : safe_downcast<int, int64_t>(padding[1]);

  TORCH_CHECK(dilation.size() == 1 || dilation.size() == 2,
    "rectify: dilation must either be a single int, or a tuple of two ints");
  const int dilationH = safe_downcast<int, int64_t>(dilation[0]);
  const int dilationW = dilation.size() == 1 ? dilationH : safe_downcast<int, int64_t>(dilation[1]);

  TORCH_CHECK((input_.ndimension() == 3 || input_.ndimension() == 4),
    "non-empty 3D or 4D (batch mode) tensor expected for input");

  const int64_t nbatch = input_.ndimension() == 4 ? input_.size(-4) : 1;
  const int64_t nInputPlane = input_.size(-3);
  const int64_t inputHeight = input_.size(-2);
  const int64_t inputWidth = input_.size(-1);

  //const int64_t outputHeight = pooling_output_shape<int64_t>(inputHeight, kH, padH, dH, dilationH, false);
  //const int64_t outputWidth = pooling_output_shape<int64_t>(inputWidth, kW, padW, dW, dilationW, false);
  const int64_t outputHeight = output.size(-2);
  const int64_t outputWidth = output.size(-1);

  pool2d_shape_check(
    input_,
    kH, kW, dH, dW, padH, padW, dilationH, dilationW,
    nInputPlane,
    inputHeight, inputWidth,
    outputHeight, outputWidth);

  at::Tensor input = input_.contiguous();

  //output.resize_({nbatch, nInputPlane, outputHeight, outputWidth});

  const int32_t count = safe_downcast<int32_t, int64_t>(output.numel());
  const uint32_t  num_threads = std::min(at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  const uint32_t num_blocks = at::cuda::ATenCeilDiv<uint32_t>(count, num_threads);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "conv_rectify_cuda_frame", ([&] {
        //using accscalar_t = acc_type<scalar_t, true>;
        scalar_t *output_data = output.data_ptr<scalar_t>();
        conv_rectify_cuda_frame<scalar_t, scalar_t>
            <<<num_blocks, num_threads, 0, at::cuda::getCurrentCUDAStream()>>>(
            count,
                nbatch,
                nInputPlane,
                inputHeight, inputWidth,
                outputHeight, outputWidth,
                kH, kW,
                dH, dW,
                padH, padW,
                dilationH, dilationW,
                output_data,
                average);
  }));


  AT_CUDA_CHECK(hipGetLastError());
}

void CONV_RECTIFY_CUDA(
  at::Tensor& output,
  const at::Tensor& input,
  at::IntArrayRef kernel_size,
  at::IntArrayRef stride,
  at::IntArrayRef padding,
  at::IntArrayRef dilation,
  bool average) {
  //at::Tensor output = at::empty({0}, input.options());
  conv_rectify_cuda_tempalte(
    output,
    input,
    kernel_size,
    stride,
    padding,
    dilation,
    average);
}


